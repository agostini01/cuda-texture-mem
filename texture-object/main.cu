#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <stdio.h>

#include <numeric>
#include <vector>
#include <iostream>
#include <iomanip>

void printArrayAsMatrix(const float* in, const size_t& width, const size_t& height) {
  for (size_t j = 0; j < height; ++j) {
    for (size_t i = 0; i < width; ++i) {
      std::cout <<std::fixed 
        << std::setw(5) // space between numbers
        << std::setprecision(2) // nubmers after decimal point
        << in[width*j + i] << ' ';
    }
    std::cout << std::endl;
  }
}

__global__ void transformKernel (float * d_output,
    hipTextureObject_t texObj,
    int width, int height,
    float theta)
{
  // Calculate normalized texture coordinates
  unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

  float u = x / (float)width;
  float v = y / (float)height;
 
  // And regular coordinates
  unsigned int idx= y * width + x;

  // Transform coordinates
  u -= 0.0f;
  v -= 0.0f;
  float tu = u * cosf(theta) - v * sinf(theta) + 0.0f;
  float tv = v * cosf(theta) + u * sinf(theta) + 0.0f;
  // Read from texture and write to global memory
  d_output[idx] = tex2D<float>(texObj, tu, tv);
  
  if(idx==10) printf("%d\n", idx);
  if(idx==10) printf("%f,%f\n", tu,tv);
  if(idx==10) printf("%f\n", d_output[idx]);
  if(idx==10) printf("%f\n", tex2D<float>(texObj, tu, tv));

}



int main () 
{
  // Inputs
  size_t width = 16;
  size_t height = 16;
  size_t size = width * height * sizeof(float);
  float angle = 0;

  // Initialize host array 
  float * h_data = (float*)malloc(size);
  for (int i =0; i<height*width; ++i) h_data[i] =(float)i/(height*width);

  // hipArray obj will have elements of 32bits, representing single-precision
  // floating point numbers
  hipChannelFormatDesc ch_desc =
    hipCreateChannelDesc(32,0,0,0,
        hipChannelFormatKindFloat);

  hipArray* cu_array;
  checkCudaErrors(hipMallocArray(&cu_array, &ch_desc, width, height));

  checkCudaErrors(hipMemcpyToArray(cu_array, 0, 0, h_data, size,
      hipMemcpyHostToDevice));

  // Specify texture
  // Texture is going to be bound to a 1D Array, with name cu_array
  struct hipResourceDesc res_desc;
  memset(&res_desc, 0, sizeof(res_desc));
  res_desc.resType = hipResourceTypeArray;
  res_desc.res.array.array = cu_array;

  // Specify texture object parameters
  // - Wrap mode: when outside of the boorderd, index x is converted to
  //   frac(x)=x floor(x) with floor(x) is the largest integer 
  //   not greater than x
  // - With interpoation
  // - No conversion/normalization of the value read
  // - Coordinates are normalized to -1,1: useful for trigonometry
  struct hipTextureDesc tex_desc;
  memset(&tex_desc, 0, sizeof(tex_desc));
  tex_desc.addressMode[0]   = hipAddressModeWrap;
  tex_desc.addressMode[1]   = hipAddressModeWrap;
  tex_desc.filterMode       = hipFilterModeLinear;
  tex_desc.readMode         = hipReadModeElementType;
  tex_desc.normalizedCoords = 1;

  // Create texture object
  hipTextureObject_t tex_obj = 0;
  hipCreateTextureObject(&tex_obj, &res_desc, &tex_desc, NULL);

  // Allocate result of transformation in device memory
  float* d_output;
  checkCudaErrors(hipMalloc(&d_output, size));
  
  // Print host array
  printArrayAsMatrix(h_data, width, height);

  // Invoke kernel
  dim3 dimBlock(16, 16);
  dim3 dimGrid((width  + dimBlock.x - 1) / dimBlock.x,
      (height + dimBlock.y - 1) / dimBlock.y);
  transformKernel<<<dimGrid, dimBlock>>>(d_output,
      tex_obj, width, height,
      angle);

  checkCudaErrors(hipMemcpy(h_data, d_output, size, hipMemcpyDeviceToHost));


  // Print host array
  printArrayAsMatrix(h_data, width, height);

  // Destroy texture object
  checkCudaErrors(hipDestroyTextureObject(tex_obj));

  // Free device memory
  checkCudaErrors(hipFreeArray(cu_array));
  checkCudaErrors(hipFree(d_output));
  // Free other
  free(h_data);
}
