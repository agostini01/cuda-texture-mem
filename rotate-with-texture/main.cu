#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <stdio.h>

#include <numeric>
#include <vector>
#include <iostream>
#include <iomanip>

#define PI_F 3.141592654f

void printArrayAsMatrix(const float* in, 
    const size_t& width, const size_t& height) {
  std::cout <<"Printing "<<width<<","<<height<<" array"<< std::endl;
  for (size_t j = 0; j < height; ++j) {
    for (size_t i = 0; i < width; ++i) {
      std::cout <<std::fixed 
        << std::setw(5) // space between numbers
        << std::setprecision(2) // nubmers after decimal point
        << in[width*j + i] << ' ';
    }
    std::cout << std::endl;
  }
}

__global__ void rotateKernel (float * output,
    hipTextureObject_t texObj, int width, int height,
    float theta) {

  // Calculate normalized texture coordinates
  unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

  float u = x / (float)width;
  float v = y / (float)height;
 
  // And regular coordinates
  unsigned int idx= y * width + x;

  // Transform coordinates
  u -= 0.5f;
  v -= 0.5f;
  float tu = u * cosf(theta) - v * sinf(theta) + 0.5f;
  float tv = v * cosf(theta) + u * sinf(theta) + 0.5f;

  // Read from texture and write to global memory
  output[idx] = tex2D<float>(texObj, tu, tv);
}

int main () 
{
  // Inputs
  size_t width = 16;
  size_t height = 16;
  size_t size = width * height * sizeof(float);
  float angle = 90; // in degrees
  float theta = angle/180*PI_F; // in rad

  // Initialize host array 
  float * h_data = (float*)malloc(size);
  for (int i =0; i<height*width; ++i) h_data[i] =(float)i/(height*width);
  memset(h_data, 0, size/4);

  // hipArray obj will have elements of 32bits, representing single-precision
  // floating point numbers
  hipChannelFormatDesc ch_desc =
    hipCreateChannelDesc(32,0,0,0,
        hipChannelFormatKindFloat);

  hipArray* cu_array;
  checkCudaErrors(hipMallocArray(&cu_array, &ch_desc, width, height));

  checkCudaErrors(hipMemcpyToArray(cu_array, 0, 0, h_data, size,
      hipMemcpyHostToDevice));

  // Specify texture
  // Texture is going to be bound to a 1D Array, with name cu_array
  struct hipResourceDesc res_desc;
  memset(&res_desc, 0, sizeof(res_desc));
  res_desc.resType = hipResourceTypeArray;
  res_desc.res.array.array = cu_array;

  // Specify texture object parameters
  // - Clamp mode: if out of bounds clamp index to closest 0 or width | 0 or height
  // - Without interpoation
  // - No conversion/normalization of the value read
  // - Coordinates are normalized to -1,1: useful for trigonometry
  struct hipTextureDesc tex_desc;
  memset(&tex_desc, 0, sizeof(tex_desc));
  tex_desc.addressMode[0]   = hipAddressModeClamp;
  tex_desc.addressMode[1]   = hipAddressModeClamp;
  tex_desc.filterMode       = hipFilterModePoint;
  tex_desc.readMode         = hipReadModeElementType;
  tex_desc.normalizedCoords = 1;

  // Create texture object
  hipTextureObject_t tex_obj = 0;
  hipCreateTextureObject(&tex_obj, &res_desc, &tex_desc, NULL);

  // Allocate result of transformation in device memory
  float* d_output;
  checkCudaErrors(hipMalloc(&d_output, size));
  
  // Print host array
  printArrayAsMatrix(h_data, width, height);

  // Invoke kernel
  dim3 dimBlock(16, 16);
  dim3 dimGrid(
      (width  + dimBlock.x - 1) / dimBlock.x,
      (height + dimBlock.y - 1) / dimBlock.y);
  rotateKernel<<<dimGrid, dimBlock>>>(d_output,
      tex_obj, width, height,
      theta);

  checkCudaErrors(hipMemcpy(h_data, d_output, size, hipMemcpyDeviceToHost));

  // Print host array
  printArrayAsMatrix(h_data, width, height);

  // Destroy texture object
  checkCudaErrors(hipDestroyTextureObject(tex_obj));

  // Free device memory
  checkCudaErrors(hipFreeArray(cu_array));
  checkCudaErrors(hipFree(d_output));

  // Free host memory
  free(h_data);
}
