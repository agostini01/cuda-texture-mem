#include "hip/hip_runtime.h"
// Code adapted from stack oveflow:
// https://stackoverflow.com/a/38749995

#include <stdio.h>
#include <hip/hip_runtime_api.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

texture<float, hipTextureType3D, hipReadModeElementType>
    volumeTexIn;
surface<void, 3> volumeTexOut;

__global__ void
surf_write(float *data, hipExtent volumeSize)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    if (x >= volumeSize.width || y >= volumeSize.height || z >= volumeSize.depth)
    {
        return;
    }

    float output = data[z * (volumeSize.width * volumeSize.height) + y * (volumeSize.width) + x];

    surf3Dwrite(output, volumeTexOut, x * sizeof(float), y, z);
}

__global__ void
tex_read(float x, float y, float z)
{
    printf("x: %f, y: %f, z:%f, val: %f\n", x, y, z, tex3D(volumeTexIn, x, y, z));
}

void runtest(float *data, hipExtent vol, float x, float y, float z)
{
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
    hipArray_t content;
    checkCudaErrors(hipMalloc3DArray(&content, &channelDesc, vol, hipArraySurfaceLoadStore));

    float *d_data;
    checkCudaErrors(hipMalloc(&d_data, vol.width * vol.height * vol.depth * sizeof(float)));
    checkCudaErrors(hipMemcpy(d_data, data, vol.width * vol.height * vol.depth * sizeof(float), hipMemcpyHostToDevice));

    // dim3 blockSize(8, 8, 8);
    dim3 gridSize((vol.width+7)/8,(vol.height+7)/8, (vol.depth+7)/8);
    volumeTexIn.filterMode = hipFilterModeLinear;

    checkCudaErrors(hipBindTextureToArray(volumeTexIn, content));
    tex_read<<<1, 1>>>(x, y, z);
    checkCudaErrors(hipDeviceSynchronize());

    hipFreeArray(content);
    hipFree(d_data);
    return;
}

int main()
{
    const int dim = 8;
    float *data = (float *)malloc(dim * dim * dim * sizeof(float));
    for (int z = 0; z < dim; z++)
        for (int y = 0; y < dim; y++)
            for (int x = 0; x < dim; x++)
            {
                data[z * dim * dim + y * dim + x] = z * 100 + y * 10 + x;
                printf("x: %f, y: %f, z:%f, val: %f\n", x, y, z, data[z * dim * dim + y * dim + x]);
            }

    hipExtent vol = {dim, dim, dim};
    runtest(data, vol, 1.5, 1.5, 1.5);
    runtest(data, vol, 1.6, 1.6, 1.6);
    runtest(data, vol, 10, 10, 10);

    free(data);
    return 0;
}