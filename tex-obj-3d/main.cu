#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <stdio.h>

#include <numeric>
#include <vector>
#include <iostream>
#include <iomanip>

void printArrayAsCharMatrix(const float* in, const size_t& width, const size_t& height) {
    std::cout << std::endl;
    char buffer[4];
    int ret;
    for (size_t j = 0; j < height; ++j)
    {
      for (size_t i = 0; i < width; ++i)
      {
        ret=snprintf(buffer, sizeof buffer, "%f", in[width*j + i]);

        // if (ret < 0) {
        //   return EXIT_FAILURE;
        // }
        // if (ret >= sizeof buffer) {
        // }
        
        std::cout << buffer[0]
                  << buffer[1]
                  << buffer[2]
                  << buffer[3]
                  << ' ';
      }
      std::cout << std::endl;
  }
}

void printArrayAsMatrix(const float* in, const size_t& width, const size_t& height) {
    std::cout << std::endl;
  for (size_t j = 0; j < height; ++j) {
    for (size_t i = 0; i < width; ++i) {
      std::cout <<std::fixed 
        << std::setw(5) // space between numbers
        << std::setprecision(2) // nubmers after decimal point
        << in[width*j + i] << ' ';
    }
    std::cout << std::endl;
  }
}

__global__ void copyKernel(hipSurfaceObject_t inputSurfObj,
                           hipSurfaceObject_t outputSurfObj,
                           int width, int height) 
{
    // Calculate surface coordinates
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int idx = y * width + x;

    if (x < width && y < height) {
        uchar4 data;
        // Read from input surface
        surf2Dread(&data,  inputSurfObj, x * 4, y);
        if(idx==100) printf("%d\n", data.w);
        // Write to output surface
        data.w+=5;
        surf2Dwrite(data, outputSurfObj, x * 4, y);
    }
}

int main () 
{
  // Inputs
  size_t width = 16;
  size_t height = 16;
  size_t size = width * height * sizeof(float);

  // Initialize host array 
  float * h_data = (float*)malloc(size);
  for (int i =0; i<height*width; ++i) h_data[i] =(float)i/(height*width);

  hipChannelFormatDesc channelDesc =
      hipCreateChannelDesc(8, 8, 8, 8,
                            hipChannelFormatKindUnsigned);

  hipArray *cuInputArray;
  hipMallocArray(&cuInputArray, &channelDesc, width, height,
                  hipArraySurfaceLoadStore);

  hipArray *cuOutputArray;
  hipMallocArray(&cuOutputArray, &channelDesc, width, height,
                  hipArraySurfaceLoadStore);

  // checkCudaErrors(hipMemcpyToArray(cu_array, 0, 0, h_data, size,
  //     hipMemcpyHostToDevice));


  // Copy to device memory some data located at address h_data
  // in host memory
  hipMemcpyToArray(cuInputArray, 0, 0, h_data, size,
                    hipMemcpyHostToDevice);

  // Specify surface
  struct hipResourceDesc resDesc;
  memset(&resDesc, 0, sizeof(resDesc));
  resDesc.resType = hipResourceTypeArray;

  // Create the surface objects
  resDesc.res.array.array = cuInputArray;
  hipSurfaceObject_t inputSurfObj = 0;
  hipCreateSurfaceObject(&inputSurfObj, &resDesc);

  resDesc.res.array.array = cuOutputArray;
  hipSurfaceObject_t outputSurfObj = 0;
  hipCreateSurfaceObject(&outputSurfObj, &resDesc);

  // Allocate output buffer in device memory
  float* d_output;
  checkCudaErrors(hipMalloc(&d_output, size));
  // Print result array
  // checkCudaErrors(hipMemcpyToArray(cu_array, 0, 0, d_output, size,
  //     hipMemcpyDeviceToDevice));
  // checkCudaErrors(hipMemcpyFromArray(d_output, cuInputArray, 0, 0, size,
  //     hipMemcpyDeviceToDevice));
  // checkCudaErrors(hipMemcpy(h_data, d_output, size, hipMemcpyDeviceToHost));
  checkCudaErrors(hipMemcpyFromArray(h_data, cuInputArray, 0, 0, size,
      hipMemcpyDeviceToHost));
  // printArrayAsMatrix(h_data, width, height);
  printArrayAsCharMatrix(h_data, width, height);


  // Invoke kernel
  dim3 dimBlock(16, 16);
  dim3 dimGrid((width  + dimBlock.x - 1) / dimBlock.x,
      (height + dimBlock.y - 1) / dimBlock.y);
  copyKernel<<<dimGrid, dimBlock>>>(inputSurfObj,
                                    outputSurfObj,
                                    width, height);
  
  // checkCudaErrors(hipMemcpyFromArray(d_output, cuOutputArray, 0, 0, size,
  //     hipMemcpyDeviceToDevice));
  // checkCudaErrors(hipMemcpy(h_data, d_output, size, hipMemcpyDeviceToHost));
  checkCudaErrors(hipMemcpyFromArray(h_data, cuOutputArray, 0, 0, size,
      hipMemcpyDeviceToHost));
  // printArrayAsMatrix(h_data, width, height);
  printArrayAsCharMatrix(h_data, width, height);

  // Destroy surface objects
  hipDestroySurfaceObject(inputSurfObj);
  hipDestroySurfaceObject(outputSurfObj);

  // Free device memory
  checkCudaErrors(hipFreeArray(cuInputArray));
  checkCudaErrors(hipFreeArray(cuOutputArray));
  checkCudaErrors(hipFree(d_output));

  // Free other
  free(h_data);

  return 0;
}

