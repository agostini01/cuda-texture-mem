#include "hip/hip_runtime.h"
// adapted from:
// The docs

#include <stdio.h>
#include <hip/hip_runtime_api.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_texture_types.h>
#include <hip/hip_fp16.h>

__global__ void transformKernel(float *output, hipTextureObject_t texObj,
                                int width, int height, float theta)
{
    unsigned x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned y = blockIdx.y * blockDim.y + threadIdx.y;
    float u = x / (float)width;
    float v = y / (float)height;

    u -= 0.5f;
    v -= 0.5f;
    float tu = u * cosf(theta) - v * sinf(theta) + 0.5f;
    float tv = v * cosf(theta) + u * sinf(theta) + 0.5f;

    // Reads from texture, writes to array
    output[y*width + x] = tex2D<float>(texObj, tu, tv);
}


__global__ void
regular_read(int x, int y, float * array, int width, int height)
{
    printf("regular: x: %d, y: %d, val: %f\n", x, y, array[x+y*width]);
}

int main()
{
    const int dim = 128;
    size_t width = dim;
    size_t height = dim;
    float angle = dim;

    // Initialize Host memory
    float *h_data = (float *)malloc(dim * dim * dim * sizeof(float));
    for (int y = 0; y < dim; y++)
        for (int x = 0; x < dim; x++)
            h_data[y * dim + x] = y * 10 + x;

    // Initialize a 2D array device array and copy host data to device
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc(
        32, 0, 0, 0, hipChannelFormatKindFloat);
    hipArray * cuArray;
    hipMallocArray(&cuArray, &channelDesc, width, height);
    hipMemcpy2DToArray(cuArray, 0, 0, h_data, width*sizeof(float), width * sizeof(float), height, hipMemcpyHostToDevice);

    // Specify the texture
    struct hipResourceDesc resDesc;
    memset(&resDesc, 0, sizeof(resDesc));
    resDesc.resType = hipResourceTypeArray;
    resDesc.res.array.array = cuArray;

    struct hipTextureDesc texDesc;
    memset(&texDesc, 0, sizeof(texDesc));
    texDesc.addressMode[0] = hipAddressModeWrap;
    texDesc.addressMode[1] = hipAddressModeWrap;
    texDesc.filterMode = hipFilterModeLinear;
    texDesc.readMode = hipReadModeElementType;
    texDesc.normalizedCoords = 1;

    hipTextureObject_t texObj = 0;
    hipCreateTextureObject(&texObj, &resDesc, &texDesc, NULL);

    // Allocates a regular array on the device
    float * output;
    hipMalloc(&output, width * height * sizeof(float));
    hipMemcpy(output,h_data,width*height*sizeof(float), hipMemcpyHostToDevice);


    // angle = 0;
    // Grid/block dims to transform the array
    dim3 dimBlock(16,16);
    dim3 dimGrid(
        (width + dimBlock.x - 1) / dimBlock.x,
        (height + dimBlock.y - 1) / dimBlock.y);
    
    regular_read<<<1, 1>>>(20,20, output, width, height);
    transformKernel<<<dimGrid, dimBlock>>>(output, texObj, width, height, angle);
    regular_read<<<1, 1>>>(20,20, output, width, height);


    // Cleanup
    hipDestroyTextureObject(texObj);
    hipFreeArray(cuArray);
    hipFree(output);
    free(h_data);
    return 0;



}